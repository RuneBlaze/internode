#include "hip/hip_runtime.h"
/*
__device__ void printBitString(unsigned int data){
  for (int j = 0;j < 32 ; j++) {
    if(j % 4 == 0 && j != 0){
      printf(" ");
    }
    printf("%d",((data  >> j) & 1));
  }
  printf("\n");
}


__device__ void printDNASequence(unsigned int data){
  for (int j = 0;j < 16; j++) {
    if(j % 4 == 0 && j != 0){
      printf(" ");
    }
    unsigned int c = ((data >> (j*2)) & 3);
    switch (c) {
    case 0: {
      printf("A ");
      break;
    }
    case 1: {
      printf("G ");
      break;
    }
    case 2: {
      printf("T ");
      break;
    }
    case 3: {
      printf("C ");
      break;
    }	  
    default:
      break;
    }
  }
  printf("\n");
}
*/

#include "cutil_inline.h"
#include "constants.hpp"
#include <sys/time.h>

__shared__ unsigned int shared_data_dna[threads_pr_block*3];

__device__ void parallelReductionDNA(){
    //wait for all threads to finish and reuse shared memory to store results
  __syncthreads();
  
  //perform parallel reduction
  for(unsigned int s=blockDim.x/2; s>32; s>>=1){    
    if (threadIdx.x < s){
      shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + s];
      shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x + s + threads_pr_block];
      shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x + s + threads_pr_block*2];
    }
    __syncthreads();
  }
  
  if (threadIdx.x < 32) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 32];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x + 32 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x + 32 + threads_pr_block*2];
  }   
  __syncthreads();
  if (threadIdx.x < 16) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 16];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x + 16 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x + 16 + threads_pr_block*2];
  }
  __syncthreads();
  if (threadIdx.x < 8) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 8];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x +  8 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x +  8 + threads_pr_block*2];
  }
  __syncthreads();
  if (threadIdx.x < 4) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 4];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x +  4 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x +  4 + threads_pr_block*2];
  }  
  __syncthreads();
  
  if (threadIdx.x < 2) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 2];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x + 2 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x + 2 + threads_pr_block*2];
  }  
  __syncthreads();

  if(threadIdx.x == 0){
    shared_data_dna[0] += shared_data_dna[1];
    shared_data_dna[0 + threads_pr_block] += shared_data_dna[1 + threads_pr_block];
    shared_data_dna[0 + threads_pr_block*2] += shared_data_dna[1 + threads_pr_block*2];
  }
  __syncthreads();
}

/*each block computes all distances between a sequence and all other sequences with higher indexes. In this way no summation is needed but shared memory is not utillised*/
__global__ void computeSliceDNA(unsigned int* gaps_gpu, unsigned int* ts_gpu, unsigned int* tv_gpu, unsigned int* bitStrings_gpu, unsigned int* gapFilters_gpu, unsigned int bsStride, unsigned int dataSize, unsigned int rowOffset){

  int i = blockIdx.y;
  if( i < blockIdx.x+1+rowOffset){
    return;
  }
  unsigned int iterations = dataSize / threads_pr_block;
  if(dataSize % threads_pr_block != 0){
    iterations++;
  }
  //for(int i = blockIdx.x+1+rowOffset; i < sequenceCount_GPU; i++) {
  //set counters in shared memory
  shared_data_dna[threadIdx.x] = 0; 
  shared_data_dna[threadIdx.x + threads_pr_block] = 0;
  shared_data_dna[threadIdx.x + threads_pr_block*2] = 0;
  for(int slice = 0; slice < iterations; slice++) {     
    // fetch next block of sequence data
    unsigned int idx0 = (rowOffset + blockIdx.x)*bsStride + slice*threads_pr_block + threadIdx.x; 
    unsigned int baseSeq = bitStrings_gpu[idx0];
    unsigned int baseGap = gapFilters_gpu[idx0];
    unsigned int idx1 = i*bsStride + slice*threads_pr_block + threadIdx.x;
    //printf("%d*%d+%d*%d+%d=%d \n",i,bsStride,slice,threads_pr_block,threadIdx.x,idx1);
    unsigned int r = bitStrings_gpu[idx1];
    unsigned int gf = gapFilters_gpu[idx1];
      
    // compute distances and gaps
    r = r ^ baseSeq;
    gf = gf & baseGap;
    unsigned int tv = (r >> 1) & 0x55555555;
    unsigned int ts = (~tv) & (r & 0x55555555);
      
    // handle gaps
    tv = tv & gf;
    ts = ts & gf;
    // sum distances in this thread
    tv = __popc(tv);
    ts = __popc(ts);
    gf = __popc(gf);
      
    //Remove invalid results
    if(slice * threads_pr_block + threadIdx.x >= dataSize) {
      tv = 0;
      ts = 0;
      gf = 0;
    }
      
    //save intermediate result
    shared_data_dna[threadIdx.x] += tv;
    shared_data_dna[threadIdx.x + threads_pr_block] += ts;
    shared_data_dna[threadIdx.x + threads_pr_block*2] += gf;    
  }

  //sum over all threads
  parallelReductionDNA();

  //store data in global memory
  if(threadIdx.x == 0){
    //printf("%d \n", blockIdx.x * sequenceCount_GPU + i);
    tv_gpu[blockIdx.x * gridDim.y + i] = shared_data_dna[0];
    ts_gpu[blockIdx.x * gridDim.y + i] = shared_data_dna[threads_pr_block];
    gaps_gpu[blockIdx.x * gridDim.y + i] = shared_data_dna[threads_pr_block*2];
  }
}

unsigned int sequenceCount_dna;
unsigned int dataSize_dna;
size_t resultMemSize_dna;
size_t bitStringsMemSize_dna;
//results
unsigned int* ts_gpu;
unsigned int* tv_gpu;
unsigned int* gaps_dna_gpu;
unsigned int bsStride_dna;
//data
unsigned int* gapFilters_dna_gpu;
unsigned int* bitStrings_dna_gpu;
unsigned int rowsPrKernel;
unsigned int numberOfKernelLaunches;

#ifdef TIMING
extern float totalGpuComputation;
extern timeval start,end;
extern float totalTransfer;
#endif

extern "C" void computeDistancesDNA_gpu(unsigned int* ts, unsigned int* tv, unsigned int* gaps) {

  // execute the kernel
  //  unsigned int startIdx = i * threads_pr_block;
  for(int i = 0; i < numberOfKernelLaunches; i++){
    printf("kernel: %d -----------------------\n",i);
    // setup execution parameters
    unsigned int gridSize_dna = 0;
    if(i != numberOfKernelLaunches-1){
      gridSize_dna = rowsPrKernel;
    } else {
      gridSize_dna = sequenceCount_dna - (numberOfKernelLaunches-1)*rowsPrKernel;
    }
    dim3 grid(gridSize_dna,sequenceCount_dna);
    dim3 block(threads_pr_block, 1, 1);
    unsigned int transferSize = gridSize_dna * sequenceCount_dna * sizeof(unsigned int);
    computeSliceDNA<<< grid, block >>>(gaps_dna_gpu, ts_gpu, tv_gpu, bitStrings_dna_gpu, gapFilters_dna_gpu, bsStride_dna, dataSize_dna, i*rowsPrKernel);
    
#ifdef TIMING
    hipDeviceSynchronize();
    gettimeofday(&end,NULL);
    totalGpuComputation += (end.tv_sec - start.tv_sec)*1000.0 + (end.tv_usec - start.tv_usec)/1000.0;
    gettimeofday(&start,NULL);
#endif

  cutilSafeCall(hipMemcpy(&ts[i * rowsPrKernel * sequenceCount_dna], ts_gpu, transferSize, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(&tv[i * rowsPrKernel * sequenceCount_dna], tv_gpu, transferSize, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(&gaps[i * rowsPrKernel * sequenceCount_dna], gaps_dna_gpu, transferSize, hipMemcpyDeviceToHost));
  
#ifdef TIMING
  hipDeviceSynchronize();
  gettimeofday(&end,NULL);
  totalTransfer += (end.tv_sec - start.tv_sec)*1000.0 + (end.tv_usec - start.tv_usec)/1000.0;
  gettimeofday(&start,NULL);
#endif
  }
}

extern "C" void getResultsDNA_gpu(unsigned int* ts, unsigned int* tv, unsigned int* gaps) {
  // Wait for kernels to finish.
  cutilCheckMsg("Kernel execution failed");  

  // Copy results from device to host
  /*cutilSafeCall(hipMemcpy(ts, ts_gpu, resultMemSize_dna, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(tv, tv_gpu, resultMemSize_dna, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(gaps, gaps_dna_gpu, resultMemSize_dna, hipMemcpyDeviceToHost));*/
}

extern "C" void storeDataDNA_gpu(unsigned int* bitStrings, unsigned int* gapFilters) {
  // Copy results from device to host
  cutilSafeCall(hipMemcpyAsync(bitStrings_dna_gpu, bitStrings, bitStringsMemSize_dna, hipMemcpyHostToDevice,0));
  cutilSafeCall(hipMemcpyAsync(gapFilters_dna_gpu, gapFilters, bitStringsMemSize_dna, hipMemcpyHostToDevice,0));
}

extern "C" void initialiseDNA_gpu(unsigned int sequenceCount, unsigned int bitStringCount, unsigned int _bsStride) {
  printf("initialising GPU... \n");
  sequenceCount_dna = sequenceCount;
  bsStride_dna = _bsStride;  
  dataSize_dna = min(bsStride_dna,bitStringCount * 4);

  //printf("TEST: %d %d %d\n",bsStride_dna, bitStringCount, dataSize_dna);
  //exit(0);
  
  
  resultMemSize_dna = sequenceCount * sequenceCount * sizeof(unsigned int);
  bitStringsMemSize_dna = bsStride_dna * sizeof(unsigned int) * sequenceCount;
  
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);  
  float freeGpuMem = deviceProp.totalGlobalMem - 250*1024*1024; //substract 100MB
  
    
  //printf("TODO REMOVE THIS \n");
  //float freeGpuMem = 50.0 * 1024 * 1024;

  freeGpuMem -= bitStringsMemSize_dna;
  float temp = min(1.0,freeGpuMem / (resultMemSize_dna*3.0));
  rowsPrKernel = sequenceCount * temp;
  numberOfKernelLaunches = sequenceCount/rowsPrKernel;
  if(sequenceCount % rowsPrKernel != 0){
    numberOfKernelLaunches++;
  }

  rowsPrKernel = sequenceCount / numberOfKernelLaunches;
  unsigned int exRows = sequenceCount - rowsPrKernel*numberOfKernelLaunches;
  rowsPrKernel += exRows;

  printf("rows pr kernel: %d\n",rowsPrKernel);
  resultMemSize_dna = rowsPrKernel * sequenceCount * sizeof(unsigned int);

  //exit(0);
  // allocate device memory
  cutilSafeCall(hipMalloc((void**) &bitStrings_dna_gpu, bitStringsMemSize_dna));
  cutilSafeCall(hipMalloc((void**) &gapFilters_dna_gpu, bitStringsMemSize_dna));
  cutilSafeCall(hipMalloc((void**) &ts_gpu, resultMemSize_dna));
  cutilSafeCall(hipMalloc((void**) &tv_gpu, resultMemSize_dna));
  cutilSafeCall(hipMalloc((void**) &gaps_dna_gpu, resultMemSize_dna));
}
