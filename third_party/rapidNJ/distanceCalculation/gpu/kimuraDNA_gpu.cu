#include "hip/hip_runtime.h"
/*
__device__ void printBitString(unsigned int data){
  for (int j = 0;j < 32 ; j++) {
    if(j % 4 == 0 && j != 0){
      printf(" ");
    }
    printf("%d",((data  >> j) & 1));
  }
  printf("\n");
}


__device__ void printDNASequence(unsigned int data){
  for (int j = 0;j < 16; j++) {
    if(j % 4 == 0 && j != 0){
      printf(" ");
    }
    unsigned int c = ((data >> (j*2)) & 3);
    switch (c) {
    case 0: {
      printf("A ");
      break;
    }
    case 1: {
      printf("G ");
      break;
    }
    case 2: {
      printf("T ");
      break;
    }
    case 3: {
      printf("C ");
      break;
    }	  
    default:
      break;
    }
  }
  printf("\n");
}
*/

#include "cutil_inline.h"
#include "constants.hpp"
#include <sys/time.h>

__shared__ unsigned int shared_data_dna[threads_pr_block*3];

__device__ void parallelReductionDNA2(){
    //wait for all threads to finish and reuse shared memory to store results
  __syncthreads();
  
  //perform parallel reduction
  for(unsigned int s=blockDim.x/2; s>32; s>>=1){    
    if (threadIdx.x < s){
      shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + s];
      shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x + s + threads_pr_block];
      shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x + s + threads_pr_block*2];
    }
    __syncthreads();
  }
  
  if (threadIdx.x < 32) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 32];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x + 32 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x + 32 + threads_pr_block*2];
  }   
  __syncthreads();
  if (threadIdx.x < 16) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 16];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x + 16 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x + 16 + threads_pr_block*2];
  }
  __syncthreads();
  if (threadIdx.x < 8) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 8];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x +  8 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x +  8 + threads_pr_block*2];
  }
  __syncthreads();
  if (threadIdx.x < 4) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 4];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x +  4 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x +  4 + threads_pr_block*2];
  }  
  __syncthreads();
  
  if (threadIdx.x < 2) {
    shared_data_dna[threadIdx.x] += shared_data_dna[threadIdx.x + 2];
    shared_data_dna[threadIdx.x + threads_pr_block] += shared_data_dna[threadIdx.x + 2 + threads_pr_block];
    shared_data_dna[threadIdx.x + threads_pr_block*2] += shared_data_dna[threadIdx.x + 2 + threads_pr_block*2];
  }  
  __syncthreads();

  if(threadIdx.x == 0){
    shared_data_dna[0] += shared_data_dna[1];
    shared_data_dna[0 + threads_pr_block] += shared_data_dna[1 + threads_pr_block];
    shared_data_dna[0 + threads_pr_block*2] += shared_data_dna[1 + threads_pr_block*2];
  }
  __syncthreads();
}

/*each block computes all distances between a sequence and all other sequences with higher indexes. In this way no summation is needed but shared memory is not utillised*/
__global__ void computeSliceDNA(float* results, unsigned int* bitStrings_gpu, unsigned int* gapFilters_gpu, unsigned int bsStride, unsigned int dataSize, unsigned int rowOffset){

  int i = blockIdx.y;
  if( i < blockIdx.x+1+rowOffset){
    return;
  }
  unsigned int iterations = dataSize / threads_pr_block;
  if(dataSize % threads_pr_block != 0){
    iterations++;
  }
  //for(int i = blockIdx.x+1+rowOffset; i < sequenceCount_GPU; i++) {
  //set counters in shared memory
  shared_data_dna[threadIdx.x] = 0; 
  shared_data_dna[threadIdx.x + threads_pr_block] = 0;
  shared_data_dna[threadIdx.x + threads_pr_block*2] = 0;
  for(int slice = 0; slice < iterations; slice++) {     
    // fetch next block of sequence data
    unsigned int idx0 = (rowOffset + blockIdx.x)*bsStride + slice*threads_pr_block + threadIdx.x; 
    unsigned int baseSeq = bitStrings_gpu[idx0];
    unsigned int baseGap = gapFilters_gpu[idx0];
    unsigned int idx1 = i*bsStride + slice*threads_pr_block + threadIdx.x;
    //printf("%d*%d+%d*%d+%d=%d \n",i,bsStride,slice,threads_pr_block,threadIdx.x,idx1);
    unsigned int r = bitStrings_gpu[idx1];
    unsigned int gf = gapFilters_gpu[idx1];
      
    // compute distances and gaps
    r = r ^ baseSeq;
    gf = gf & baseGap;
    unsigned int tv = (r >> 1) & 0x55555555;
    unsigned int ts = (~tv) & (r & 0x55555555);
      
    // handle gaps
    tv = tv & gf;
    ts = ts & gf;
    // sum distances in this thread
    tv = __popc(tv);
    ts = __popc(ts);
    gf = __popc(gf);
      
    //Remove invalid results
    if(slice * threads_pr_block + threadIdx.x >= dataSize) {
      tv = 0;
      ts = 0;
      gf = 0;
    }
      
    //save intermediate result
    shared_data_dna[threadIdx.x] += tv;
    shared_data_dna[threadIdx.x + threads_pr_block] += ts;
    shared_data_dna[threadIdx.x + threads_pr_block*2] += gf;    
  }

  //sum over all threads
  parallelReductionDNA2();

  //store data in global memory
  if(threadIdx.x == 0){
    //printf("%d \n", blockIdx.x * sequenceCount_GPU + i);
    //tv_gpu[blockIdx.x * gridDim.y + i] = shared_data_dna[0];
    //ts_gpu[blockIdx.x * gridDim.y + i] = shared_data_dna[threads_pr_block];
    unsigned int lengthWithoutGaps = shared_data_dna[threads_pr_block*2];
    float ts = float(shared_data_dna[threads_pr_block]) / lengthWithoutGaps;
    float tv = float(shared_data_dna[0]) / lengthWithoutGaps;
    float temp1 = 1.0f-2.0f*ts-tv;
    float temp2 = 1.0f-2.0f*tv;
    float distance = -1.0f;
    if(!(temp1 <= 0 || temp2 <= 0)){
      distance = -0.5f*log(1.0f-2.0f*ts-tv)-0.25f*log(1.0f-2.0f*tv);
    }
    results[blockIdx.x * gridDim.y + i] = distance;
  }
}

unsigned int sequenceCount2_dna;
unsigned int dataSize2_dna;
size_t resultMemSize2_dna;
size_t bitStringsMemSize2_dna;

//results
float* results_dna_gpu;
unsigned int bsStride2_dna;

//data
unsigned int* gapFilters2_dna_gpu;
unsigned int* bitStrings2_dna_gpu;
unsigned int rowsPrKernel2;
unsigned int numberOfKernelLaunches2;

#ifdef TIMING
extern float totalGpuComputation;
extern timeval start,end;
extern float totalTransfer;
#endif

extern "C" void computeDistancesDNA2_gpu(float* results) {

  // execute the kernel
  //  unsigned int startIdx = i * threads_pr_block;
  for(int i = 0; i < numberOfKernelLaunches2; i++) {
    printf("kernel: %d -----------------------\n",i);
    // setup execution parameters
    unsigned int gridSize_dna = 0;
    if(i != numberOfKernelLaunches2-1){
      gridSize_dna = rowsPrKernel2;
    } else {
      gridSize_dna = sequenceCount2_dna - (numberOfKernelLaunches2-1)*rowsPrKernel2;
    }
    dim3 grid(gridSize_dna,sequenceCount2_dna);
    dim3 block(threads_pr_block, 1, 1);
    unsigned int transferSize = gridSize_dna * sequenceCount2_dna * sizeof(unsigned int);
    computeSliceDNA<<< grid, block >>>(results_dna_gpu, bitStrings2_dna_gpu, gapFilters2_dna_gpu, bsStride2_dna, dataSize2_dna, i*rowsPrKernel2);
    
#ifdef TIMING
    hipDeviceSynchronize();
    gettimeofday(&end,NULL);
    totalGpuComputation += (end.tv_sec - start.tv_sec)*1000.0 + (end.tv_usec - start.tv_usec)/1000.0;
    gettimeofday(&start,NULL);
#endif

    cutilSafeCall(hipMemcpy(&results[i * rowsPrKernel2 * sequenceCount2_dna], results_dna_gpu, transferSize, hipMemcpyDeviceToHost));
  
#ifdef TIMING
  hipDeviceSynchronize();
  gettimeofday(&end,NULL);
  totalTransfer += (end.tv_sec - start.tv_sec)*1000.0 + (end.tv_usec - start.tv_usec)/1000.0;
  gettimeofday(&start,NULL);
#endif
  }
}

extern "C" void storeDataDNA2_gpu(unsigned int* bitStrings, unsigned int* gapFilters) {
  // Copy results from device to host
  cutilSafeCall(hipMemcpyAsync(bitStrings2_dna_gpu, bitStrings, bitStringsMemSize2_dna, hipMemcpyHostToDevice,0));
  cutilSafeCall(hipMemcpyAsync(gapFilters2_dna_gpu, gapFilters, bitStringsMemSize2_dna, hipMemcpyHostToDevice,0));
}

extern "C" void initialiseDNA2_gpu(unsigned int sequenceCount, unsigned int bitStringCount, unsigned int _bsStride) {
  printf("initialising GPU... \n");
  sequenceCount2_dna = sequenceCount;
  bsStride2_dna = _bsStride;
  dataSize2_dna = min(bsStride2_dna,bitStringCount * 4);

  resultMemSize2_dna = sequenceCount * sequenceCount * sizeof(float);
  bitStringsMemSize2_dna = bsStride2_dna * sizeof(unsigned int) * sequenceCount;
  
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);  
  float gpuMemSize = deviceProp.totalGlobalMem * 0.8f;
  
  printf("Gpu memsize %fMB\n",gpuMemSize/1024.0f/1024.0f);

  //printf("TODO REMOVE THIS \n");
  //float gpuMemSize = 50.0 * 1024 * 1024;

  
  rowsPrKernel2 = sequenceCount * min(1.0f,(gpuMemSize - bitStringsMemSize2_dna*2) / resultMemSize2_dna);
  numberOfKernelLaunches2 = sequenceCount/rowsPrKernel2;
  if(sequenceCount % rowsPrKernel2 != 0){
    numberOfKernelLaunches2++;
  }
  rowsPrKernel2 = sequenceCount / numberOfKernelLaunches2;
  while(sequenceCount > rowsPrKernel2*numberOfKernelLaunches2){
    rowsPrKernel2++;
  }
  printf("rows pr kernel: %d\n",rowsPrKernel2);
  resultMemSize2_dna = rowsPrKernel2 * sequenceCount * sizeof(unsigned int);

  // allocate device memory
  cutilSafeCall(hipMalloc((void**) &bitStrings2_dna_gpu, bitStringsMemSize2_dna));
  cutilSafeCall(hipMalloc((void**) &gapFilters2_dna_gpu, bitStringsMemSize2_dna));
  cutilSafeCall(hipMalloc((void**) &results_dna_gpu, resultMemSize2_dna));
}
