#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include "constants.hpp"
#include <math.h>
#include <sys/time.h>

__shared__ unsigned int shared_data_protein[threads_pr_block];

__device__ void parallelReductionProtein(unsigned int val){
  //wait for all threads to finish and use shared memory to store results
  shared_data_protein[threadIdx.x] = val;
  __syncthreads();

   //perform parallel reduction
  for(unsigned int s=blockDim.x/2; s>32; s>>=1){    
    if (threadIdx.x < s){
      shared_data_protein[threadIdx.x ] += shared_data_protein[threadIdx.x + s];
    } 
    __syncthreads();
  }
  if (threadIdx.x < 32) {
    shared_data_protein[threadIdx.x ] += shared_data_protein[threadIdx.x + 32 ];
  }   
  __syncthreads();
  

  if (threadIdx.x < 16) {
    shared_data_protein[threadIdx.x ] += shared_data_protein[threadIdx.x + 16 ];
  }
  __syncthreads();
  if (threadIdx.x < 8) {
    shared_data_protein[threadIdx.x ] += shared_data_protein[threadIdx.x +  8 ];
  }
  __syncthreads();

  if (threadIdx.x < 4) {
    shared_data_protein[threadIdx.x ] += shared_data_protein[threadIdx.x +  4 ];
  }  
  __syncthreads();
  
  if (threadIdx.x < 2) {
    shared_data_protein[threadIdx.x ] += shared_data_protein[threadIdx.x + 2 ];
  }  
  __syncthreads();

  if(threadIdx.x == 0){
    shared_data_protein[0 ] += shared_data_protein[1 ];
  }
  __syncthreads();
}

__global__ void computeDistanceProtein(unsigned int* gaps_gpu, unsigned int* dist_gpu, unsigned int* bitStrings_gpu, unsigned int bsStride, unsigned int rowOffset, unsigned int dataSize_protein){
    
  int seq2 = blockIdx.y;
  if(seq2 < blockIdx.x+1+rowOffset){
    return;
  }

  uchar4* charData = (uchar4*) bitStrings_gpu;
  unsigned int iterations = dataSize_protein / threads_pr_block;
  if(dataSize_protein % threads_pr_block != 0){
    iterations++;
  }
  //printf(" %d \n",iterations);
  //for(int seq2 = blockIdx.x+1+rowOffset; seq2 < sequenceCount_gpu; seq2++) {
  unsigned int sumR = 0;
  unsigned int sumG = 0;
  for(int slice = 0; slice < iterations; slice++) {      
    uchar4 r, g;
    // fetch next block of sequence data
    unsigned int idx0 = (rowOffset+blockIdx.x)*bsStride + slice*threads_pr_block + threadIdx.x;
    uchar4 base = charData[idx0];
    unsigned int idx1 = seq2*bsStride + slice*threads_pr_block + threadIdx.x;
    uchar4 target = charData[idx1];
    
    r.x = base.x != target.x;
    g.x = base.x < 64 || target.x < 64;
    r.x = r.x && !g.x;
    
    r.y = base.y != target.y;
    g.y = base.y < 64 || target.y < 64;
    r.y = r.y && !g.y;
    
    r.z = base.z != target.z;
    g.z = base.z < 64 || target.z < 64;
    r.z = r.z && !g.z;
    
    r.w = base.w != target.w;
    g.w = base.w < 64 || target.w < 64;
    r.w = r.w && !g.w;
    
    if(slice * threads_pr_block + threadIdx.x < dataSize_protein) {
      sumR += r.x;
      sumR += r.y;
      sumR += r.z;
      sumR += r.w;
      
      sumG += g.x;
      sumG += g.y;
      sumG += g.z;
      sumG += g.w;
      //printf("%d: %d %d: %d %d %d %d\n",bsStride,idx0,idx1, g.x, g.y, g.z, g.w);
    }
  }
  //sum over all threads
  parallelReductionProtein(sumR);
  sumR = shared_data_protein[0];
  parallelReductionProtein(sumG);
  sumG = shared_data_protein[0];
  
  //store result in global memory
  if(threadIdx.x == 0) {
    dist_gpu[blockIdx.x * gridDim.y + seq2] = sumR;
    gaps_gpu[blockIdx.x * gridDim.y + seq2] = sumG;
  }
}

//-------------------------------------------------------------------------------------------------------------

unsigned int gridSize_protein;
size_t resultMemSize_protein;
size_t bitStringsMemSize_protein;
unsigned int sequenceCount;
//results
unsigned int* dist_protein_gpu;
unsigned int* gaps_protein_gpu;
unsigned int bsStride_protein;
//data
unsigned int* bitStrings_protein_gpu;
unsigned int numberOfKernelLaunches_protein;
unsigned int rowsPrKernel_protein;
unsigned int dataSize_protein;

#ifdef TIMING
extern float totalGpuComputation;
extern timeval start,end;
extern float totalTransfer;
#endif

extern "C" void computeDistancesProtein_gpu(unsigned int* bitStrings, unsigned int* dist, unsigned int* gaps) {
  //  printf("%p %p %d \n",bitStrings_protein_gpu,bitStrings,bitStringsMemSize_protein);
  cutilSafeCall(hipMemcpyAsync(bitStrings_protein_gpu, bitStrings, bitStringsMemSize_protein, hipMemcpyHostToDevice,0));
  for(int i = 0; i < numberOfKernelLaunches_protein; i++){
    printf("kernel: %d -----------------------\n",i);
    // setup execution parameters
    unsigned int gridSize = 0;
    if(i != numberOfKernelLaunches_protein-1){
      gridSize = rowsPrKernel_protein;
    } else {
      gridSize = sequenceCount - (numberOfKernelLaunches_protein-1)*rowsPrKernel_protein;
    }
    printf("gridSize: %d %d\n",gridSize,sequenceCount);
    dim3 grid(gridSize,sequenceCount);
    dim3 block(threads_pr_block, 1, 1);
    unsigned int transferSize = gridSize * sequenceCount * sizeof(unsigned int);
    computeDistanceProtein<<< grid, block >>>(gaps_protein_gpu, dist_protein_gpu, bitStrings_protein_gpu, bsStride_protein, i*rowsPrKernel_protein, dataSize_protein);

#ifdef TIMING
    hipDeviceSynchronize();
    gettimeofday(&end,NULL);
    totalGpuComputation += (end.tv_sec - start.tv_sec)*1000.0 + (end.tv_usec - start.tv_usec)/1000.0;
    gettimeofday(&start,NULL);
#endif

    printf("Copying results...\n");
    cutilSafeCall(hipMemcpy(&dist[i * rowsPrKernel_protein * sequenceCount], dist_protein_gpu, transferSize, hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(&gaps[i * rowsPrKernel_protein * sequenceCount], gaps_protein_gpu, transferSize, hipMemcpyDeviceToHost));
    printf("Finished\n");

#ifdef TIMING
    hipDeviceSynchronize();
    gettimeofday(&end,NULL);
    totalTransfer += (end.tv_sec - start.tv_sec)*1000.0 + (end.tv_usec - start.tv_usec)/1000.0;
    gettimeofday(&start,NULL);
#endif
  }
}

extern "C" void getResultsProtein_gpu(unsigned int* dist, unsigned int* gaps) {
  /*  // Wait for kernels to finish.
  cutilCheckMsg("Kernel execution failed");  
  
  // Copy results from device to host
  cutilSafeCall(hipMemcpy(dist, dist_protein_gpu, resultMemSize_protein, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(gaps, gaps_protein_gpu, resultMemSize_protein, hipMemcpyDeviceToHost));
  */
  printf("getResult is not implemented for proteins \n");
  exit(1);
}

extern "C" void initialiseProtein_gpu(unsigned int _sequenceCount, unsigned int _bsStride, unsigned int paddedLength) {
  //  printf("initialising GPU... \n");
  bsStride_protein = _bsStride;
  sequenceCount = _sequenceCount;
  dataSize_protein = paddedLength / 4;  
  //printf("%d %d \n",bsStride_protein,paddedLength);
  //exit(0);

  resultMemSize_protein = sequenceCount * sequenceCount * sizeof(unsigned int);
  bitStringsMemSize_protein = bsStride_protein * sizeof(unsigned int) * sequenceCount;
  
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);  
  float freeGpuMem = deviceProp.totalGlobalMem - 250*1024*1024; //substract 100MB
    
  freeGpuMem -= bitStringsMemSize_protein;
  float temp = min(1.0,freeGpuMem / (resultMemSize_protein*2.0));
  rowsPrKernel_protein = sequenceCount * temp;

  //rowsPrKernel_protein = 500;
  
  numberOfKernelLaunches_protein = sequenceCount/rowsPrKernel_protein;
  if(sequenceCount % rowsPrKernel_protein != 0){
    numberOfKernelLaunches_protein++;
  }
  
  printf("Number of kernel launches needed: %d\n",numberOfKernelLaunches_protein);
  
  rowsPrKernel_protein = sequenceCount / numberOfKernelLaunches_protein;
  unsigned int exRows = sequenceCount - rowsPrKernel_protein*numberOfKernelLaunches_protein;
  rowsPrKernel_protein += exRows;
  
  printf("rows pr kernel: %d\n",rowsPrKernel_protein);
  resultMemSize_protein = rowsPrKernel_protein * sequenceCount * sizeof(unsigned int);
  
  // allocate device memory
  cutilSafeCall(hipMalloc((void**) &bitStrings_protein_gpu, bitStringsMemSize_protein)); 
  cutilSafeCall(hipMalloc((void**) &dist_protein_gpu, resultMemSize_protein));
  cutilSafeCall(hipMalloc((void**) &gaps_protein_gpu, resultMemSize_protein));
}
